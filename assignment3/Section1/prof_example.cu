
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 256


// Kernel function to add two vectors
__global__ void add(int *a, int *b, int *c, size_t num) {
    int block_start = blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    int index = block_start + thread_id;
    if (index < num) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    size_t num = 100000;

    int * host_a = new int[num];
    int * host_b = new int[num];
    int * host_c = new int[num];

    // Initialize host arrays
    for (int i = 0; i < num; i++) {
        host_a[i] = i;
        host_b[i] = i;
    }
    

    // Allocate memory on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, num * sizeof(int));
    hipMalloc((void**)&d_b, num * sizeof(int));
    hipMalloc((void**)&d_c, num * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, host_a, num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, host_b, num * sizeof(int), hipMemcpyHostToDevice);

    dim3 num_block((num + BLOCKSIZE - 1) / BLOCKSIZE);
    dim3 num_threads(BLOCKSIZE);

    
    int warm_up_count = 100;
    int profile_count = 100;
    size_t L2_size = 50 * 1024 * 1024;

    for (int i = 0; i < warm_up_count; ++i)
    {
        add<<<num_block, num_threads>>>(d_a, d_b, d_c, num);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    
    int* clear_l2_buffer;
    hipMalloc(&clear_l2_buffer, L2_size);

    float total_ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    for (int i = 0; i < profile_count; ++i)
    {
        hipMemset(clear_l2_buffer, 0, L2_size); // Clear L2 cache https://github.com/NVIDIA/nvbench/blob/main/nvbench/detail/l2flush.cuh
        hipEventRecord(start);
        add<<<num_block, num_threads>>>(d_a, d_b, d_c, num);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_ms += ms;
    }
    
    float average_time = total_ms / profile_count; 
    std::cout << "Average time: " << average_time << " ms" << std::endl;

    // Free the L2 buffer
    hipFree(clear_l2_buffer);
    // Free CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;

    return 0;
}

