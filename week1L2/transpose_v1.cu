#include <hip/hip_runtime.h>
#include <stdio.h>
//nvcc transpose_v1.cu -arch=compute_90 -code=sm_90
//ncu --set full python torch_transpose.py
__global__ void transpose(float *A, float *B, int num_row, int num_column) {
    int row = blockIdx.x;
    int col_start = threadIdx.x * blockDim.x;
    int col_end = col_start + blockDim.x;

    for (int col = col_start; col < col_end && col < num_column; col++) {
        B[col * num_row + row] = A[row * num_column + col];
    }
}

int main(){
    int num_row = 8192;
    int num_column = 8192;
    float * host_A, * host_B;
    float * device_A, * device_B;

    host_A = (float *)malloc(num_row * num_column * sizeof(float));
    host_B = (float *)malloc(num_row * num_column * sizeof(float));
    hipMalloc((void**)&device_A, num_row * num_column * sizeof(float));
    hipMalloc((void**)&device_B, num_row * num_column * sizeof(float));

    // Initialize host_A with some values
    for (int i = 0; i < num_row * num_column; i++) {
        host_A[i] = static_cast<float>(i);
    }
    // Copy host_A to device_A
    hipMemcpy(device_A, host_A, num_row * num_column * sizeof(float), hipMemcpyHostToDevice);
    // Launch kernel to transpose matrix
    dim3 grid(num_row);
    dim3 block(128);
    transpose<<<grid, block>>>(device_A, device_B, num_row, num_column);
    // Copy transposed matrix back to host_B
    hipMemcpy(host_B, device_B, num_row * num_column * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_A);
    hipFree(device_B);

    // check the result
    for (int i = 0; i < num_row; i++) {
        for (int j = 0; j < num_column; j++) {
            if (host_B[i * num_column + j] != host_A[j * num_row + i]) {
                printf("Error at (%d, %d): %f != %f\n", i, j, host_B[i * num_column + j], host_A[j * num_row + i]);
                return -1;
            }
        }
    }
    printf("Transpose successful!\n");
}