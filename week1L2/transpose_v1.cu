#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void transposeKernel(float* input, float* output, int num_rows, int num_cols) {
    int row = blockIdx.x;
    int col_start = threadIdx.x * num_cols / blockDim.x;
    int col_end = (threadIdx.x + 1) * num_cols / blockDim.x;

    for (int col = col_start; col < col_end; col++) {
        if (col < num_cols) {
            output[col * num_rows + row] = input[row * num_cols + col];
        }
    }
}

int main(){
    int num_rows = 8192;
    int num_cols = 8192;

    float *h_input, *h_output;
    float *d_input, *d_output;

    size_t size = num_rows * num_cols * sizeof(float);
    h_input = (float*)malloc(size);
    h_output = (float*)malloc(size);
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Initialize input matrix
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            h_input[i * num_cols + j] = static_cast<float>(i * num_cols + j);
        }
    }

    // Copy input matrix to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 gridDim(num_rows);
    dim3 blockDim(128);
    transposeKernel<<<gridDim, blockDim>>>(d_input, d_output, num_rows, num_cols);
    hipDeviceSynchronize();

    // Copy output matrix back to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    // Verify the result
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            if (h_output[j * num_rows + i] != h_input[i * num_cols + j]) {
                printf("Mismatch at (%d, %d): %f != %f\n", i, j, h_output[j * num_rows + i], h_input[i * num_cols + j]);
                break;
            }
        }
    }
    printf("Transpose completed successfully.\n");
    
    // Free memory
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}