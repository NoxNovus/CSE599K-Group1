#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 256


// Kernel function to add two vectors
__global__ void add(int *a, int *b, int *c, size_t num) {
    int block_start = blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    int index = block_start + thread_id;
    if (index < num) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    size_t num = 1000000000;

    int * host_a = new int[num];
    int * host_b = new int[num];
    int * host_c = new int[num];

    // Initialize host arrays
    for (int i = 0; i < num; i++) {
        host_a[i] = i;
        host_b[i] = i;
    }
    

    // Allocate memory on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, num * sizeof(int));
    hipMalloc((void**)&d_b, num * sizeof(int));
    hipMalloc((void**)&d_c, num * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, host_a, num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, host_b, num * sizeof(int), hipMemcpyHostToDevice);

    dim3 num_block((num + BLOCKSIZE - 1) / BLOCKSIZE);
    dim3 num_threads(BLOCKSIZE);
    /*
    The below code corresponds to the cudaLaunchKernel signature

    __host__​cudaError_t cudaLaunchKernel ( const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, cudaStream_t stream )

    num_block corresponds to gridDim: how many blocks you want to launch.
    num_threads corresponds to blockDim: how many threads per block you want.

    */
    add<<<num_block, num_threads>>>(d_a, d_b, d_c, num);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Copy result back to host
    hipMemcpy(host_c, d_c, num * sizeof(int), hipMemcpyDeviceToHost);


    for (int i = 0; i < num; i++) {
        if (host_c[i] != host_a[i] + host_b[i]) {
            std::cerr << "Error at index " << i << ": " << host_c[i] << std::endl;
            break;
        }
    }

    std::cout << "Result: " << host_c[0] << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;

    return 0;
}