#include<hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>

void initVector(float* vector, int size) {
    srand(1);
    for (int i = 0; i < size; i++) {
        vector[i] = (float)rand() / RAND_MAX;
    }
}

float measureTime(float* host_vector, int size) {
    float* device_vector;
    hipMalloc((void**)&device_vector, size * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // start timing
    hipEventRecord(start);

    // memory transfers
    // cudaMemcpy(device_vector, host_vector, size * sizeof(float), cudaMemcpyHostToDevice);
    hipMemcpy(host_vector, device_vector, size * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    hipFree(device_vector);
    // Print results
    return time;
}

int main(){
    // warmup
    for (int i = 0; i < 10; i++) {
        int warmupSize = 1024;
        float* host_warmupVector = (float *) malloc(warmupSize * sizeof(float));
        initVector(host_warmupVector, warmupSize);
        float* device_warmupVector;
        hipMalloc((void**)&device_warmupVector, warmupSize * sizeof(float));

        hipMemcpy(device_warmupVector, host_warmupVector, warmupSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(host_warmupVector, device_warmupVector, warmupSize * sizeof(float), hipMemcpyDeviceToHost);

        free(host_warmupVector);
        hipFree(device_warmupVector);
    }

    // iterations
    bool pinnedMemory = false;
    int num_bytes = 1;
    int num_iter = 25;
    float times[num_iter];
    for (int i = 0; i < num_iter; i ++) {
        float *vector;
        if (pinnedMemory) {
            hipHostMalloc(&vector, num_bytes * sizeof(float), hipHostMallocDefault);
        } else {
            vector = (float *) malloc(num_bytes * sizeof(float));
        }
        initVector(vector, num_bytes);
        times[i] = measureTime(vector, num_bytes);
        num_bytes *= 2;
        if (pinnedMemory) {
            hipHostFree(vector);
        } else {
            free(vector);
        }
    }
    // graph bandwidth vs transfer size curve
    num_bytes = 1;
    for (int i = 0; i < num_iter; i ++) {
        printf("Transfer Size: %d \n", num_bytes);
        printf("Bandwidth: %f \n", num_bytes / times[i]);

        num_bytes *= 2;
    }
}